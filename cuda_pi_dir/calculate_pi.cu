#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <limits>

#include <hip/hip_runtime.h>

#include "timer.hpp"

using namespace std;

__global__ void calcpi(int threads, int n, double *results) {
   int rank = threadIdx.x;
   results[rank] = 0.0;
   double step = 1.0/n;
   double x = 0.0;

   int lower = rank * n/threads;
   int upper = (rank + 1) * n/threads;

   for (long i = lower; i < upper; i++) {
      x    = (i + 0.5) * step;
      results[rank] += 4.0 / (1.0 + x*x);
   }
}

int main( int argc, char **argv ) {
   long num_steps = 100000;
   double result;
   int threads = 100; // threads needs to dived num_steps!

   cout.precision(numeric_limits<double>::digits10+2);
   
   if (argc > 1) {
      num_steps = atol(argv[1]);
   }
   if (argc > 2) {
      threads = atol(argv[2]);
   }

   double step, pi;
   Timer timer;
   
   cout << "Calculating PI using:" << endl <<
           "  " << num_steps << " slices" << endl <<
           "  " << threads << " CUDA threads" << endl;
   
   timer.start();
   
   double *sum, *d_sum;
   size_t size = threads*sizeof(double);
   step = 1.0 / num_steps;
   sum = (double*)malloc(size);

   hipMalloc((void**)&d_sum, size);
   calcpi<<<1,threads>>>(threads, num_steps, d_sum);
   hipMemcpy(sum, d_sum, size, hipMemcpyDeviceToHost);
   hipFree(d_sum);

   result = 0.0;

   for (int i=0; i<threads; i++) {
      result +=sum[i];
   }
   pi = result * step;

   timer.stop();

   cout << "Obtained value for PI: " << pi << endl <<
           "Time taken: " << timer.duration() << " seconds" << endl;

   return 0;
}

